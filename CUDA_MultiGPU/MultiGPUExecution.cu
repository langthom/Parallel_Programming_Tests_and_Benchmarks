#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2023 Dr. Thomas Lang
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
**/

#include <algorithm>
#include <cassert>
#include <numeric>
#include "MultiGPUExecution.h"
#include "../Common/CommonKernels.h"

#ifdef HAS_CUDA

#include "../Common/StatisticsKernel.h"


double getMaxAllocationSizeMultiCUDAGPU(double maxMemoryInGiB, double actuallyUsePercentage) {
  constexpr size_t toGiB = 1ull << 30;
  size_t maxMemoryInBytes = static_cast< size_t >(maxMemoryInGiB * toGiB);

  // Detect all available CUDA devices.
  int nr_gpus;
  std::vector< std::string > deviceNames;
  std::vector< size_t > availableMemoryPerDevice, totalMemoryPerDevice;
  hipError_t error = getGPUInformation(nr_gpus, deviceNames, availableMemoryPerDevice, totalMemoryPerDevice);

  if (error == hipError_t::hipSuccess) {
    // Compute the total size available on all devices.
    size_t totalAvailableOnAllDevices = std::accumulate(availableMemoryPerDevice.cbegin(), availableMemoryPerDevice.cend(), 0ull);
    maxMemoryInBytes = std::min< size_t >(maxMemoryInBytes, totalAvailableOnAllDevices);
  }

  double maxMemoryInGB = static_cast< double >(maxMemoryInBytes) / toGiB;
  maxMemoryInGB *= actuallyUsePercentage;
  return maxMemoryInGB;
}


std::vector< std::pair< std::size_t, std::size_t > > partitionVolumeForMultiGPU(int& nr_gpus, hipError_t& error, int K, std::size_t dimZ) {
  std::vector< double > memoryFractionsPerDevice;
  {
    std::vector< std::string > deviceNames;
    std::vector< size_t > availableMemoryPerDevice, totalMemoryPerDevice;
    error = getGPUInformation(nr_gpus, deviceNames, availableMemoryPerDevice, totalMemoryPerDevice);
    if(error != hipError_t::hipSuccess) {
      return {};
    }

    size_t totalMemoryAvailable = std::accumulate(availableMemoryPerDevice.cbegin(), availableMemoryPerDevice.cend(), 0ull);

    memoryFractionsPerDevice.resize(nr_gpus);
    std::transform(availableMemoryPerDevice.cbegin(), availableMemoryPerDevice.cend(), memoryFractionsPerDevice.begin(),
                  [totalMemoryAvailable](size_t availablePerDevice) { return static_cast< double >(availablePerDevice) / totalMemoryAvailable; });
  }

  std::vector< std::size_t > chunkZs(nr_gpus);
  chunkZs[0] = 0;
  for(int gpuID = 1; gpuID < nr_gpus; ++gpuID) {
    chunkZs[gpuID] = chunkZs[gpuID-1] + static_cast< std::size_t >(static_cast< double >(dimZ) * memoryFractionsPerDevice[gpuID-1]);
  }


  std::vector< std::pair< std::size_t, std::size_t > > partitions;
  partitions.reserve(nr_gpus);
  std::size_t K2 = K / 2ull;

  for(int gpuID = 0; gpuID < nr_gpus; ++gpuID) {
    std::size_t begin = gpuID     == 0       ? 0ull     : chunkZs[gpuID]     - K2;
    std::size_t end   = gpuID + 1 == nr_gpus ? dimZ - 1 : chunkZs[gpuID + 1] + K2 - 1;
    partitions.emplace_back(begin, end);
  }

  return partitions;
}


hipError_t launchKernelMultiCUDAGPU(float* out, float* in, int64_t N, int64_t* offsets, int K, int64_t dimX, int64_t dimY, int64_t dimZ, float* elapsedTime, int threadsPerBlock) {
#define HANDLE_ERROR(err)             if(error != hipError_t::hipSuccess) { return error; }
#define HANDLE_ERROR_STMT(err, stmts) if(error != hipError_t::hipSuccess) { stmts; return error; }

  assert(out); assert(in); assert(offsets); assert(elapsedTime);

  hipError_t error = hipError_t::hipSuccess;
  int nr_gpus = 0;

  // 1. Compute the partitioning over the devices, i.e., how many slices of the given block are processed
  //    on which device. The computed indices are z-slice indices which are inclusive on both sides.
  auto partitioning = partitionVolumeForMultiGPU(nr_gpus, error, K, dimZ);
  HANDLE_ERROR(error);

  // 2. Create a stream for each device and events for synchronization/timing.
  std::vector< hipStream_t > streams(nr_gpus);
  std::vector< hipEvent_t > timingEvents(2 * nr_gpus);
  for(int gpuID = 0; gpuID < nr_gpus; ++gpuID) {
    hipSetDevice(gpuID);
    error = hipStreamCreate(&streams[gpuID]);
    error = hipEventCreateWithFlags(&timingEvents[2*gpuID+0], hipEventBlockingSync);
    error = hipEventCreateWithFlags(&timingEvents[2*gpuID+1], hipEventBlockingSync);
  }
  HANDLE_ERROR(error);
  

  // 3. Memory allocation on the devices.
  //    Note that on older devices, the asynchronous API (i.e., hipMallocAsync and its similar functions)
  //    may not be supported. Thus, we rely on the "older" way of doing things, namely doing the *blocking*
  //    calls, but concurrently on the CPU via regular CPU threads.

  std::vector< int64_t >  gpuDataAllocationSizes(nr_gpus * 2);
  std::vector< float* >   gpuDataAllocations(nr_gpus * 2);
  std::vector< int64_t* > gpuOffsetAllocations(nr_gpus);

  auto freeAllocations = [nr_gpus,&gpuDataAllocations,&gpuOffsetAllocations](bool sync = true) {
    if(sync) {
      for(int gpuID = 0; gpuID < nr_gpus; ++gpuID) {
        hipSetDevice(gpuID);
        hipDeviceSynchronize();
      }
    }
    for (auto& allocPtr : gpuDataAllocations) {
      hipFree(allocPtr);
    }
    for (auto& allocPtr : gpuOffsetAllocations) {
      hipFree(allocPtr);
    }
  };

  int padding                     = K - 1;
  int64_t offsetBytes             = K * K * K * sizeof(int64_t);
  int64_t sliceSize               = dimX * dimY;
  int64_t sliceSizeWithoutPadding = (dimX - padding) * (dimY - padding);

  #pragma omp parallel num_threads(nr_gpus)
  {
    hipError_t threadLocal_error = hipError_t::hipSuccess;

    #pragma omp parallel for
    for(int gpuID = 0; gpuID < nr_gpus; ++gpuID) {
      // Change the current device.
      threadLocal_error = hipSetDevice(gpuID);

      int inputIndex  = 2 * gpuID + 0;
      int outputIndex = 2 * gpuID + 1;

      // Allocate memory for the data chunks (input and output) on the device.
      auto const& zPartition = partitioning[gpuID];
      int64_t zRange = zPartition.second - zPartition.first + 1ull; /* +1 since the upper z boundary is inclusive */
      gpuDataAllocationSizes[ inputIndex] = zRange * sliceSize * sizeof(float);
      gpuDataAllocationSizes[outputIndex] = (zRange - padding) * sliceSizeWithoutPadding * sizeof(float);

      threadLocal_error = hipMalloc(reinterpret_cast< void** >(&gpuDataAllocations[ inputIndex]), gpuDataAllocationSizes[ inputIndex]);
      threadLocal_error = hipMalloc(reinterpret_cast< void** >(&gpuDataAllocations[outputIndex]), gpuDataAllocationSizes[outputIndex]);
      threadLocal_error = hipMalloc(reinterpret_cast< void** >(&gpuOffsetAllocations[gpuID]), offsetBytes);

      #pragma omp atomic
      error = threadLocal_error;
    }
  }

  // explicit (CPU) synchronization for error handling, breaking from an openmp loop is ... weird
  HANDLE_ERROR_STMT(error, freeAllocations());
  
  for(int gpuID = 0; gpuID < nr_gpus; ++gpuID) {
    int inputIndex  = 2 * gpuID + 0;
    int outputIndex = 2 * gpuID + 1;

    hipStream_t& currentStream = streams[gpuID];

    error = hipEventRecord(timingEvents[inputIndex], currentStream);
    HANDLE_ERROR_STMT(error, freeAllocations());

    error = hipSetDevice(gpuID);
    HANDLE_ERROR_STMT(error, freeAllocations());

    float*   deviceDataIn  = gpuDataAllocations[ inputIndex];
    float*   deviceDataOut = gpuDataAllocations[outputIndex];
    int64_t* deviceOffsets = gpuOffsetAllocations[gpuID];

    int64_t inputBytes  = gpuDataAllocationSizes[ inputIndex];
    int64_t outputBytes = gpuDataAllocationSizes[outputIndex];

    auto const& zPartition = partitioning[gpuID];
    int partitionDimZ = static_cast< int >(zPartition.second - zPartition.first + 1ull);
    float* dataInBegin  =  in + zPartition.first * sliceSize;
    float* dataOutBegin = out + zPartition.first * sliceSizeWithoutPadding;

    error = hipMemcpyAsync(deviceDataIn,  dataInBegin, inputBytes,  hipMemcpyHostToDevice, currentStream);
    error = hipMemcpyAsync(deviceOffsets, offsets,     offsetBytes, hipMemcpyHostToDevice, currentStream);
    HANDLE_ERROR_STMT(error, freeAllocations());

    int64_t voxelsInCurrentPartition = partitionDimZ * sliceSize;
    int64_t blocksPerGrid = (voxelsInCurrentPartition + threadsPerBlock - 1) / threadsPerBlock;
    statisticsKernel<<< blocksPerGrid, threadsPerBlock, 0, currentStream >>>(deviceDataIn, deviceDataOut, deviceOffsets, K, voxelsInCurrentPartition, dimX, dimY, partitionDimZ);

    error = hipMemcpyAsync(dataOutBegin, deviceDataOut, outputBytes, hipMemcpyDeviceToHost, currentStream);
    HANDLE_ERROR_STMT(error, freeAllocations());
  }

  float averageElapsedTimeMs = 0.f;
  #pragma omp parallel for num_threads(nr_gpus)
  for(int gpuID = 0; gpuID < nr_gpus; ++gpuID) {
    float elapsedMs = 0.f;
    hipError_t threadLocalError = hipError_t::hipSuccess;
    hipEvent_t& stopEvent = timingEvents[2*gpuID+1];
    threadLocalError = hipEventRecord(stopEvent, streams[gpuID]);
    threadLocalError = hipEventSynchronize(stopEvent);
    threadLocalError = hipEventElapsedTime(&elapsedMs, timingEvents[2*gpuID+0], stopEvent);
    threadLocalError = hipStreamDestroy(streams[gpuID]);

    #pragma omp critical
    {
      averageElapsedTimeMs += elapsedMs;
      error = threadLocalError;
    }
  }
  *elapsedTime = averageElapsedTimeMs / nr_gpus;
  HANDLE_ERROR_STMT(error, freeAllocations());

  freeAllocations(/*sync=*/false);
  return hipError_t::hipSuccess;

#undef HANDLE_ERROR
#undef HANDLE_ERROR_STMT
}

#endif // HAS_CUDA
